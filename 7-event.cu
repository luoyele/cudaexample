#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipEvent_t start, stop;
    float elapsedTime;

    // 创建事件
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 记录开始时间
    hipEventRecord(start, 0);

    // 执行内核函数（这里用一个假设的函数名代替）
    myKernel<<<gridSize, blockSize>>>(...);

    // 记录结束时间
    hipEventRecord(stop, 0);

    // 同步事件，确保结束时间已经记录
    hipEventSynchronize(stop);

    // 计算经过时间
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Elapsed Time: " << elapsedTime << " ms" << std::endl;

    // 销毁事件
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}