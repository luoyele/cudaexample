#include<iostream>
#include<hip/hip_runtime.h>

// ref: https://blog.csdn.net/qq_17239003/article/details/78991567

using namespace std;

#define CHECK(call)                                                         \
    do                                                                      \
    {                                                                       \
        const hipError_t error_code = call;                                \
        if (error_code != hipSuccess)                                      \
        {                                                                   \
            printf("CUDA Error\n");                                         \
            printf("    File:   %s\n", __FILE__);                           \
            printf("    Line:   %d\n", __LINE__);                           \
            printf("    Error code: %d\n", error_code);                     \
            printf("    Error text: %s\n", hipGetErrorString(error_code)); \
            exit(1);                                                        \
        }                                                                   \
    } while (0)


// GPU Kernel func, perform element-wise add
__global__ void kernel_sum(int *arr1, int *arr2, int *out, int N)
{
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < N)
    {
        out[thread_id] = arr1[thread_id] + arr2[thread_id];
    }
}

int main() {
    const int N = 2048 * 2048;
    int *arr1 = new int[N];
    int *arr2 = new int[N];
    int *out = new int[N];
    srand(123456);
    for (int i = 0; i < N; i++)
    {
        arr1[i] = rand() * 5 % 255;
        arr2[i] = rand() % 128 + 5;
    }

    // 1. GPU端申请显存
    int *d_arr1 = nullptr;
    int *d_arr2 = nullptr;
    int *d_out = nullptr;
    CHECK(hipMalloc((void **)&d_arr1, sizeof(int) * N));
    CHECK(hipMalloc((void **)&d_arr2, sizeof(int) * N));
    CHECK(hipMalloc((void **)&d_out, sizeof(int) * N));

    // 2. CPU Memory数据复制到GPU显存
 
    hipMemcpy(d_arr1, arr1, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_arr2, arr2, sizeof(int) * N, hipMemcpyHostToDevice);

    // 3. 设置GPU端线程执行配置, launch the GPU kernel
    hipEvent_t start, end;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&end));
    // 开始计时
    hipEventRecord(start);

    int blk_size = 128;
    int grid_size = (N + blk_size - 1) / blk_size;
    kernel_sum<<<grid_size, blk_size>>>(d_arr1, d_arr2, d_out, N);

    // 结束计时
    hipEventRecord(end);
    hipEventSynchronize(end);
    // 统计时间
    float time_ms = 0.f;
    hipEventElapsedTime(&time_ms, start, end);
    std::cout << "CUDA Kernel time: " << time_ms << " ms" << std::endl;

    
    // 4. Cpoy GPU result to CPU
    hipMemcpy(out, d_out, sizeof(int) * N, hipMemcpyDeviceToHost);
   
    // 5. Free GPU Memory
    hipFree(d_arr1);
    hipFree(d_arr2);
    hipFree(d_out);
}